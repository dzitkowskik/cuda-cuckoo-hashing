#include "hip/hip_runtime.h"
/*
 *  fast_cuckoo_hash.hpp
 *
 *  Created on: 01-06-2015
 *      Author: Karol Dzitkowski
 *
 *  >>  Real-time Parallel Hashing on the GPU
 *
 *  Implementation of a fast cuckoo hashing method introduced in publication:
 *
 *  Dan A. Alcantara, Andrei Sharf, Fatemeh Abbasinejad, Shubhabrata Sengupta,
 *  Michael Mitzenmacher, John D. Owens, and Nina Amenta "Real-time Parallel
 *  Hashing on the GPU", ACM Transactions on Graphics
 *  (Proceedings of ACM SIGGRAPH Asia 2009)
 *
 *  which can be found here http://idav.ucdavis.edu/~dfalcant/research/hashing.php
 */

#include "fast_cuckoo_hash.cuh"
#include "hash_function.cuh"
#include "helpers.h"
#include "macros.h"
#include <thrust/scan.h>
#include <hip/hip_runtime_api.h>

__global__ void divideKernel(
		const int2* values,
		const int size,
		const Constants<2> constants,
		unsigned int* counts,
		const int bucket_cnt,
		unsigned int* offsets,
		const unsigned int max_size,
		bool* failure)
{
	unsigned idx = threadIdx.x + blockIdx.x * blockDim.x +
			blockIdx.y * blockDim.x * gridDim.x;

	if(idx >= size) return;

	int key = values[idx].x;
	unsigned hash = bucketHashFunction(
			constants.values[0], constants.values[1], key, bucket_cnt);
	offsets[idx] = atomicAdd(&counts[hash], 1);
	if(offsets[idx] == max_size - 1) *failure = true;
}

__global__ void copyKernel(
		const int2* values,
		const int size,
		const Constants<2> constants,
		unsigned int* starts,
		const int bucket_cnt,
		unsigned int* offsets,
		int2* buffer)
{
	unsigned idx = threadIdx.x + blockIdx.x * blockDim.x +
			blockIdx.y * blockDim.x * gridDim.x;

	if(idx >= size) return;

	int key = values[idx].x;
	unsigned hash = bucketHashFunction(
			constants.values[0], constants.values[1], key, bucket_cnt);
	unsigned point = starts[hash] + offsets[idx];
	buffer[point] = values[idx];
}

bool splitToBuckets(
		const int2* values,
		const int size,
		const Constants<2> constants,
		const int bucket_cnt,
		const int block_size,
		unsigned int* starts,
		unsigned int* counts,
		int2* result)
{
	auto grid = CuckooHash<2>::GetGrid(size);
	int blockSize = CuckooHash<2>::DEFAULT_BLOCK_SIZE;

	bool h_failure;
	bool* d_failure;
	unsigned int* d_offsets;

	CUDA_CALL( hipMalloc((void**)&d_offsets, size*sizeof(unsigned int)) );
	CUDA_CALL( hipMemset(d_offsets, 0, size*sizeof(unsigned int)) );

	CUDA_CALL( hipMalloc((void**)&d_failure, sizeof(bool)) );
	CUDA_CALL( hipMemset(d_failure, 0, sizeof(bool)) );

	divideKernel<<<grid, blockSize>>>(
			values, size, constants, counts,
			bucket_cnt, d_offsets, block_size, d_failure);

	hipDeviceSynchronize();
	CUDA_CALL( hipMemcpy(&h_failure, d_failure, sizeof(bool), hipMemcpyDeviceToHost) );
	CUDA_CALL( hipFree(d_failure) );

	if(h_failure == false)
	{
		thrust::device_ptr<unsigned int> starts_ptr(starts);
		thrust::device_ptr<unsigned int> counts_ptr(counts);
		auto end = thrust::exclusive_scan(counts_ptr, counts_ptr+bucket_cnt, starts_ptr);

		copyKernel<<<grid, blockSize>>>(
				values, size, constants, starts, bucket_cnt, d_offsets, result);
		hipDeviceSynchronize();
	}

	CUDA_CALL( hipFree(d_offsets) );
	return !h_failure;
}

__global__ void insertKernel(
		const int2* valuesArray,
		const unsigned int* starts,
		const unsigned int* counts,
		const int arrId,
		int2* hashMap,
		const int bucket_size,
		Constants<3> constants,
		const unsigned max_iters,
		bool* failure)
{
	volatile unsigned i, hash;
	unsigned idx = threadIdx.x;
	unsigned idx2 = threadIdx.x + blockDim.x;
	__shared__ int2 s[PART_HASH_MAP_SIZE];

	// GET DATA
	const int2* values = valuesArray + starts[arrId];
	const int size = counts[arrId];
	volatile const int part = bucket_size * arrId;
	int2* hashMap_part = hashMap + part;
	int2 old_value;
	int2 value;
	if(idx < size) value = values[idx];

	// COPY HASH MAP TO SHARED MEMORY
	s[idx] = hashMap_part[idx];
	if(idx2 < bucket_size)
	{
		s[idx2] = hashMap_part[idx2];
	}
	__syncthreads();
	bool a = true;
	#pragma unroll
	for(i = 0; a && (i<=MAX_RETRIES); i++)
	{
		if(idx < size)
		{
			hash = hashFunction(constants.values[i%3], value.x, bucket_size);
			old_value = s[hash];	// read old value
		}
		__syncthreads();
		if(idx < size) s[hash] = value;			// write new value
		__syncthreads();
		if(idx < size && value.x == s[hash].x)	// check for success
		{
			if(old_value.x == EMPTY_BUCKET_KEY)
			{
				a = false;
			}
			else
			{
				value = old_value;
			}
		}
	}
	if(idx < size && a)
	{
		*failure = true;
	}

	// COPY SHARED MEMORY TO HASH MAP
	__syncthreads();
	if(idx2 < bucket_size)
		hashMap_part[idx2] = s[idx2];
	hashMap_part[idx] = s[idx];
	__syncthreads();
}

bool fast_cuckooHash(
		const int2* values,
		const int in_size,
		int2* hashMap,
		const int bucket_cnt,
		Constants<2> bucket_constants,
		Constants<3> constants,
		int max_iters)
{
	const int block_size = FAST_CUCKOO_HASH_BLOCK_SIZE;
	unsigned int* starts;
	unsigned int* counts;
	int2* buckets;
	bool* d_failure;
	bool h_failure;

	// CREATE STREAMS
	hipStream_t* streams = new hipStream_t[bucket_cnt];
	for(int i=0; i<bucket_cnt; i++)
		CUDA_CALL( hipStreamCreate(&streams[i]) );

	// ALLOCATE MEMORY
	CUDA_CALL( hipMalloc((void**)&starts, bucket_cnt*sizeof(unsigned int)) );
	CUDA_CALL( hipMemset(starts, 0, bucket_cnt*sizeof(unsigned int)) );

	CUDA_CALL( hipMalloc((void**)&counts, bucket_cnt*sizeof(unsigned int)) );
	CUDA_CALL( hipMemset(counts, 0, bucket_cnt*sizeof(unsigned int)) );

	CUDA_CALL( hipMalloc((void**)&buckets, in_size*sizeof(int2)) );
	CUDA_CALL( hipMemset(buckets, 0xff, in_size*sizeof(int2)) );

	CUDA_CALL( hipMalloc((void**)&d_failure, sizeof(bool)) );
	CUDA_CALL( hipMemset(d_failure, 0, sizeof(bool)) );

	bool splitResult = splitToBuckets(
			values, in_size, bucket_constants, bucket_cnt,
			block_size, starts, counts, buckets);

	if(splitResult)
	{
		const int shared_mem_size = PART_HASH_MAP_SIZE * sizeof(int2);
		for(int i=0; i<bucket_cnt; i++)
		{
			insertKernel<<<1, block_size, 0, streams[i]>>>(
					values, starts, counts, i, hashMap,
					PART_HASH_MAP_SIZE, constants, max_iters, d_failure);
		}
		hipDeviceSynchronize();
		CUDA_CALL( hipMemcpy(&h_failure, d_failure, sizeof(bool), hipMemcpyDeviceToHost) );
	} else return true;

	// FREE MEMORY
	CUDA_CALL( hipFree(starts) );
	CUDA_CALL( hipFree(counts) );
	CUDA_CALL( hipFree(buckets) );
	CUDA_CALL( hipFree(d_failure) );
	for(int i=0; i<bucket_cnt; i++) CUDA_CALL( hipStreamDestroy(streams[i]) );
	delete [] streams;

	return h_failure;
}

__global__ void toInt2Kernel(const int* keys, const int size, int2* out)
{
	unsigned idx = threadIdx.x + blockIdx.x * blockDim.x +
				blockIdx.y * blockDim.x * gridDim.x;

	if(idx >= size) return;
	out[idx].x = keys[idx];
	out[idx].y = idx; // SAVE OLD POSITION
}

__global__ void retrieveKernel(
		int2* buckets,
		int2* hashMap,
		const unsigned int* starts,
		const unsigned int* counts,
		const int arrId,
		const int bucket_size,
		Constants<3> constants,
		int2* out)
{
	unsigned idx = threadIdx.x;
	const int size = counts[arrId];
	if(idx >= size) return;

	// GET DATA
	const int2* values = buckets + starts[arrId];
	const int2 value = values[idx];
	volatile int part = bucket_size * arrId;
	int2* hashMap_part = hashMap + part;
	int2 entry;
	volatile unsigned hash;

	//#pragma unroll
	for(int i = 0; i < 3; i++)
	{
		hash = hashFunction(constants.values[i], value.x, bucket_size);
		entry = hashMap_part[hash];
		if(entry.x == value.x) break;
	}

	if(entry.x == value.x)
		out[value.y] = entry;
	else
		out[value.y] = int2{EMPTY_BUCKET_KEY, EMPTY_BUCKET_KEY};

	// PLACE IT ON OLD POSITION
//	out[value.y] = entry;
//	if(entry.x == EMPTY_BUCKET_KEY)

}

int2* fast_cuckooRetrieve(
		const int* keys,
		const int size,
		int2* hashMap,
		const int bucket_cnt,
		const Constants<2> bucket_constants,
		const Constants<3> constants)
{
	auto grid = CuckooHash<2>::GetGrid(size);
	int blockSize = CuckooHash<2>::DEFAULT_BLOCK_SIZE;
	const int block_size = FAST_CUCKOO_HASH_BLOCK_SIZE;

	// ALLOCATE MEMORY
	int2 *result, *buckets;
	unsigned int *starts, *counts;
	CUDA_CALL( hipMalloc((void**)&result, size*sizeof(int2)) );
	CUDA_CALL( hipMemset(result, 0xff, size*sizeof(int2)) );

	CUDA_CALL( hipMalloc((void**)&buckets, size*sizeof(int2)) );
	CUDA_CALL( hipMemset(buckets, 0xff, size*sizeof(int2)) );

	CUDA_CALL( hipMalloc((void**)&starts, bucket_cnt*sizeof(unsigned int)) );
	CUDA_CALL( hipMemset(starts, 0, bucket_cnt*sizeof(unsigned int)) );

	CUDA_CALL( hipMalloc((void**)&counts, bucket_cnt*sizeof(unsigned int)) );
	CUDA_CALL( hipMemset(counts, 0, bucket_cnt*sizeof(unsigned int)) );

	// CREATE STREAMS
	hipStream_t* streams = new hipStream_t[bucket_cnt];
	for(int i=0; i<bucket_cnt; i++)
		CUDA_CALL( hipStreamCreate(&streams[i]) );

	// SPLIT TO BUCKETS
	toInt2Kernel<<<grid, blockSize>>>(keys, size, result);
	hipDeviceSynchronize();
	bool splitResult = splitToBuckets(
			result, size, bucket_constants, bucket_cnt, block_size, starts, counts, buckets);
	CUDA_CALL( hipMemset(result, 0xff, size*sizeof(int2)) );

	// RETRIEVE VALUES
	if(splitResult)
	{
		for(int i=0; i<bucket_cnt; i++)
		{
			retrieveKernel<<<1, block_size, 0, streams[i]>>>(
					buckets, hashMap, starts, counts, i, PART_HASH_MAP_SIZE, constants, result);
		}
		hipDeviceSynchronize();
	}
	// FREE MEMORY
	CUDA_CALL( hipFree(starts) );
	CUDA_CALL( hipFree(counts) );
	CUDA_CALL( hipFree(buckets) );
	for(int i=0; i<bucket_cnt; i++) CUDA_CALL( hipStreamDestroy(streams[i]) );
	delete [] streams;

	return result;
}
