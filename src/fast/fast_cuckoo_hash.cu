#include "hip/hip_runtime.h"
/*
 *  fast_cuckoo_hash.hpp
 *
 *  Created on: 01-06-2015
 *      Author: Karol Dzitkowski
 *
 *  >>  Real-time Parallel Hashing on the GPU
 *
 *  Implementation of a fast cuckoo hashing method introduced in publication:
 *
 *  Dan A. Alcantara, Andrei Sharf, Fatemeh Abbasinejad, Shubhabrata Sengupta,
 *  Michael Mitzenmacher, John D. Owens, and Nina Amenta "Real-time Parallel
 *  Hashing on the GPU", ACM Transactions on Graphics
 *  (Proceedings of ACM SIGGRAPH Asia 2009)
 *
 *  which can be found here http://idav.ucdavis.edu/~dfalcant/research/hashing.php
 */

#include "fast_cuckoo_hash.cuh"
#include "hash_function.cuh"
#include "helpers.h"
#include "macros.h"
#include <thrust/scan.h>
#include <hip/hip_runtime_api.h>
#include "helpers.h"

__global__ void divideKernel(
		const int2* values,
		const int size,
		const Constants<2> constants,
		unsigned int* counts,
		const int bucket_cnt,
		unsigned int* offsets,
		const unsigned int max_size,
		bool* failure)
{
	unsigned idx = threadIdx.x + blockIdx.x * blockDim.x +
			blockIdx.y * blockDim.x * gridDim.x;

	if(idx >= size) return;

	int key = values[idx].x;
	unsigned hash = bucketHashFunction(constants.values[0], constants.values[1], key, bucket_cnt);
	offsets[idx] = atomicAdd(&counts[hash], 1);
	if(offsets[idx] == max_size - 1) *failure = true;
}

__global__ void copyKernel(
		const int2* values,
		const int size,
		const Constants<2> constants,
		unsigned int* starts,
		const int bucket_cnt,
		unsigned int* offsets,
		int2* buffer)
{
	unsigned idx = threadIdx.x + blockIdx.x * blockDim.x +
			blockIdx.y * blockDim.x * gridDim.x;

	if(idx >= size) return;

	int key = values[idx].x;
	unsigned hash = bucketHashFunction(constants.values[0], constants.values[1], key, bucket_cnt);
	unsigned point = starts[hash] + offsets[idx];
	buffer[point] = values[idx];
}

bool splitToBuckets(
		const int2* values,
		const int size,
		const Constants<2> constants,
		const int bucket_cnt,
		const int block_size,
		unsigned int* starts,
		unsigned int* counts,
		int2* result)
{
	auto grid = CuckooHash<2>::GetGrid(size);
	int blockSize = CuckooHash<2>::DEFAULT_BLOCK_SIZE;

	bool h_failure;
	bool* d_failure;
	unsigned int* d_offsets;

	CUDA_CALL( hipMalloc((void**)&d_offsets, size*sizeof(unsigned int)) );
	CUDA_CALL( hipMemset(d_offsets, 0, size*sizeof(unsigned int)) );

	CUDA_CALL( hipMalloc((void**)&d_failure, sizeof(bool)) );
	CUDA_CALL( hipMemset(d_failure, 0, sizeof(bool)) );

	divideKernel<<<grid, blockSize>>>(
			values, size, constants, counts,
			bucket_cnt, d_offsets, block_size, d_failure);

	hipDeviceSynchronize();
	CUDA_CALL( hipMemcpy(&h_failure, d_failure, sizeof(bool), hipMemcpyDeviceToHost) );
	CUDA_CALL( hipFree(d_failure) );

	if(h_failure == false)
	{
		thrust::device_ptr<unsigned int> starts_ptr(starts);
		thrust::device_ptr<unsigned int> counts_ptr(counts);
		auto end = thrust::exclusive_scan(counts_ptr, counts_ptr+bucket_cnt, starts_ptr);

		copyKernel<<<grid, blockSize>>>(
				values, size, constants, starts, bucket_cnt, d_offsets, result);
		hipDeviceSynchronize();
	}

	CUDA_CALL( hipFree(d_offsets) );
	return !h_failure;
}

__global__ void insertKernel(
		const int2* valuesArray,
		const unsigned int* starts,
		const unsigned int* counts,
		const int arrId,
		int2* hashMap,
		Constants<3> constants,
		int* failures)
{
	unsigned i, hash;
	unsigned idx = threadIdx.x;
	unsigned idx2 = threadIdx.x + blockDim.x;
	__shared__ int2 s[PART_HASH_MAP_SIZE];

	// GET DATA
	const int2* values = valuesArray + starts[arrId];
	const int size = counts[arrId];
	const int part = PART_HASH_MAP_SIZE * arrId;
	int2* hashMap_part = hashMap + part;

	// COPY HASH MAP TO SHARED MEMORY
	s[idx] = hashMap_part[idx];
	if(idx2 < PART_HASH_MAP_SIZE) s[idx2] = hashMap_part[idx2];
	__syncthreads();

	int2 old_value, value;
	bool working = idx < size;
	if(working) value = values[idx];

	#pragma unroll
	for(i = 0; i <= MAX_RETRIES; i++)
	{
		hash = hashFunction(constants.values[i%3], value.x, PART_HASH_MAP_SIZE);
		old_value = s[hash];				// read old value
		__syncthreads();
		if(working) s[hash] = value;		// write new value
		__syncthreads();
		if(working && value.x == s[hash].x)		// check for success
		{
			if(value.y != s[hash].y)
				s[hash] = int2{EMPTY_BUCKET_KEY, EMPTY_BUCKET_KEY};
			else if(old_value.x == EMPTY_BUCKET_KEY)
				working = false;
			else value = old_value;
		}
	}
	if(working) atomicAdd(failures, 1);

	// COPY SHARED MEMORY TO HASH MAP
	__syncthreads();
	if(idx2 < PART_HASH_MAP_SIZE) hashMap_part[idx2] = s[idx2];
	hashMap_part[idx] = s[idx];
}

bool fast_cuckooHash(
		const int2* values,
		const int in_size,
		int2* hashMap,
		const int bucket_cnt,
		Constants<2> bucket_constants,
		Constants<3> constants,
		int max_iters)
{
	const int block_size = FAST_CUCKOO_HASH_BLOCK_SIZE;
	unsigned int* starts;
	unsigned int* counts;
	int2* buckets;
	int* d_failure;
	int h_failure;
	const int steam_no = bucket_cnt < MAX_STEAM_NO ? bucket_cnt : MAX_STEAM_NO;

	// CREATE STREAMS
	hipStream_t* streams = new hipStream_t[steam_no];
	for(int i=0; i<steam_no; i++)
		CUDA_CALL( hipStreamCreate(&streams[i]) );

	// ALLOCATE MEMORY
	CUDA_CALL( hipMalloc((void**)&starts, bucket_cnt*sizeof(unsigned int)) );
	CUDA_CALL( hipMemset(starts, 0, bucket_cnt*sizeof(unsigned int)) );

	CUDA_CALL( hipMalloc((void**)&counts, bucket_cnt*sizeof(unsigned int)) );
	CUDA_CALL( hipMemset(counts, 0, bucket_cnt*sizeof(unsigned int)) );

	CUDA_CALL( hipMalloc((void**)&buckets, in_size*sizeof(int2)) );
	CUDA_CALL( hipMemset(buckets, 0xff, in_size*sizeof(int2)) );

	CUDA_CALL( hipMalloc((void**)&d_failure, sizeof(int)) );
	CUDA_CALL( hipMemset(d_failure, 0, sizeof(int)) );

	bool splitResult = splitToBuckets(
			values, in_size, bucket_constants, bucket_cnt,
			block_size, starts, counts, buckets);

	if(splitResult)
	{
		for(int i=0; i<bucket_cnt; i++)
		{
			insertKernel<<<1, block_size, 0, streams[i%steam_no]>>>(
					buckets, starts, counts, i, hashMap, constants, d_failure);
		}
		hipDeviceSynchronize();
		CUDA_CALL( hipMemcpy(&h_failure, d_failure, sizeof(int), hipMemcpyDeviceToHost) );
	} else return true;

	// FREE MEMORY
	CUDA_CALL( hipFree(starts) );
	CUDA_CALL( hipFree(counts) );
	CUDA_CALL( hipFree(buckets) );
	CUDA_CALL( hipFree(d_failure) );
	for(int i=0; i<steam_no; i++)
		CUDA_CALL( hipStreamDestroy(streams[i]) );
	delete streams;

	return h_failure;
}

__global__ void toInt2Kernel(const int* keys, const int size, int2* out)
{
	unsigned idx = threadIdx.x + blockIdx.x * blockDim.x +
				blockIdx.y * blockDim.x * gridDim.x;

	if(idx >= size) return;
	out[idx].x = keys[idx];
	out[idx].y = EMPTY_BUCKET_KEY;
}

__global__ void retrieveKernel(
		int2* values,
		int2* hashMap,
		int size,
		int bucket_cnt,
		Constants<3> constants,
		Constants<2> bucket_constants)
{
	unsigned idx = threadIdx.x + blockIdx.x * blockDim.x +
					blockIdx.y * blockDim.x * gridDim.x;
	if(idx >= size) return;

	int key = values[idx].x;

    unsigned hash = bucketHashFunction(
			bucket_constants.values[0], bucket_constants.values[1], key, bucket_cnt);
    const unsigned bucket_start = hash * PART_HASH_MAP_SIZE;
    int2 entry;

	#pragma unroll
	for(int i = 0; i < 3; i++)
	{
		if(entry.x != key)
		{
			hash = hashFunction(constants.values[i%3], key, PART_HASH_MAP_SIZE);
			entry = hashMap[hash + bucket_start];
		}
	}

	if(entry.x == key)
		values[idx] = entry;
	else values[idx] = int2{-1,-1};
}

int2* fast_cuckooRetrieve(
		const int* keys,
		const int size,
		int2* hashMap,
		const int bucket_cnt,
		const Constants<2> bucket_constants,
		const Constants<3> constants)
{
	auto grid = CuckooHash<2>::GetGrid(size);
	int blockSize = CuckooHash<2>::DEFAULT_BLOCK_SIZE;

	// ALLOCATE MEMORY
	int2 *result;
	CUDA_CALL( hipMalloc((void**)&result, size*sizeof(int2)) );
	CUDA_CALL( hipMemset(result, 0xff, size*sizeof(int2)) );

	// SPLIT TO BUCKETS
	toInt2Kernel<<<grid, blockSize>>>(keys, size, result);
	hipDeviceSynchronize();

	retrieveKernel<<<grid, blockSize>>>(
			result, hashMap, size, bucket_cnt, constants, bucket_constants);
	hipDeviceSynchronize();

	return result;
}
