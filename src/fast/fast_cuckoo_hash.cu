#include "hip/hip_runtime.h"
/*
 *  fast_cuckoo_hash.hpp
 *
 *  Created on: 01-06-2015
 *      Author: Karol Dzitkowski
 *
 *  >>  Real-time Parallel Hashing on the GPU
 *
 *  Implementation of a fast cuckoo hashing method introduced in publication:
 *
 *  Dan A. Alcantara, Andrei Sharf, Fatemeh Abbasinejad, Shubhabrata Sengupta,
 *  Michael Mitzenmacher, John D. Owens, and Nina Amenta "Real-time Parallel
 *  Hashing on the GPU", ACM Transactions on Graphics
 *  (Proceedings of ACM SIGGRAPH Asia 2009)
 *
 *  which can be found here http://idav.ucdavis.edu/~dfalcant/research/hashing.php
 */


//
//Przeanalizowac problem dobierania funkcji hashujacej (wartosci losowej) pod wzgl�dem ewolucyjnym,
//zmieniamy pewne bity wartosci wylosowanej zamiast losowac now� i sprawdzamy czy pomog�o

#include "fast_cuckoo_hash.cuh"
#include "hash_function.cuh"
#include "helpers.h"
#include "macros.h"
#include <thrust/scan.h>
#include <hip/hip_runtime_api.h>
#include "helpers.h"

__global__ void divideKernel(
		const int2* values,
		const int size,
		const Constants<2> constants,
		unsigned int* counts,
		const int bucket_cnt,
		unsigned int* offsets,
		const unsigned int max_size,
		bool* failure)
{
	unsigned idx = threadIdx.x + blockIdx.x * blockDim.x +
			blockIdx.y * blockDim.x * gridDim.x;

	if(idx >= size) return;

	int key = values[idx].x;
	unsigned hash = bucketHashFunction(constants.values[0], constants.values[1], key, bucket_cnt);
	offsets[idx] = atomicAdd(&counts[hash], 1);
	if(offsets[idx] == max_size - 1) *failure = true;
}

__global__ void copyKernel(
		const int2* values,
		const int size,
		const Constants<2> constants,
		unsigned int* starts,
		const int bucket_cnt,
		unsigned int* offsets,
		int2* buffer)
{
	unsigned idx = threadIdx.x + blockIdx.x * blockDim.x +
			blockIdx.y * blockDim.x * gridDim.x;

	if(idx >= size) return;

	int key = values[idx].x;
	unsigned hash = bucketHashFunction(constants.values[0], constants.values[1], key, bucket_cnt);
	unsigned point = starts[hash] + offsets[idx];
	buffer[point] = values[idx];
}

bool splitToBuckets(
		const int2* values,
		const int size,
		const Constants<2> constants,
		const int bucket_cnt,
		const int block_size,
		unsigned int* starts,
		unsigned int* counts,
		int2* result)
{
	auto grid = CuckooHash<2>::GetGrid(size);
	int blockSize = CuckooHash<2>::DEFAULT_BLOCK_SIZE;

	bool h_failure;
	bool* d_failure;
	unsigned int* d_offsets;

	CUDA_CALL( hipMalloc((void**)&d_offsets, size*sizeof(unsigned int)) );
	CUDA_CALL( hipMemset(d_offsets, 0, size*sizeof(unsigned int)) );

	CUDA_CALL( hipMalloc((void**)&d_failure, sizeof(bool)) );
	CUDA_CALL( hipMemset(d_failure, 0, sizeof(bool)) );

	divideKernel<<<grid, blockSize>>>(
			values, size, constants, counts,
			bucket_cnt, d_offsets, block_size, d_failure);

	hipDeviceSynchronize();
	CUDA_CALL( hipMemcpy(&h_failure, d_failure, sizeof(bool), hipMemcpyDeviceToHost) );
	CUDA_CALL( hipFree(d_failure) );

	if(h_failure == false)
	{
		thrust::device_ptr<unsigned int> starts_ptr(starts);
		thrust::device_ptr<unsigned int> counts_ptr(counts);
		auto end = thrust::exclusive_scan(counts_ptr, counts_ptr+bucket_cnt, starts_ptr);

		copyKernel<<<grid, blockSize>>>(
				values, size, constants, starts, bucket_cnt, d_offsets, result);
		hipDeviceSynchronize();
	}

	CUDA_CALL( hipFree(d_offsets) );
	return !h_failure;
}

__global__ void insertKernel(
		const int2* valuesArray,
		const unsigned int* starts,
		const unsigned int* counts,
		const int arrId,
		int2* hashMap,
		int2* stash,
		int* stashCnt,
		Constants<3> constants,
		int* failures)
{
	unsigned i, hash, hash_idx;
	unsigned idx = threadIdx.x;
	unsigned idx2 = threadIdx.x + blockDim.x;
	__shared__ int2 s[PART_HASH_MAP_SIZE+1];

	// GET DATA
	const int2* values = valuesArray + starts[arrId];
	const int size = counts[arrId];
	const int part = PART_HASH_MAP_SIZE * arrId;
	int2* hashMap_part = hashMap + part;

	// COPY HASH MAP TO SHARED MEMORY
	s[idx] = hashMap_part[idx];
	if(idx2 < PART_HASH_MAP_SIZE)
		s[idx2] = hashMap_part[idx2];
	__syncthreads();

	int2 old_value = int2{EMPTY_BUCKET_KEY, EMPTY_BUCKET_KEY};
	bool working = idx < size;
	int2 value = working ? values[idx] : int2{EMPTY_BUCKET_KEY, EMPTY_BUCKET_KEY};

	#pragma unroll
	for(i = 0; i <= MAX_RETRIES; i++)
	{
		hash = hashFunction(constants.values[i%3], value.x, PIECE_SIZE);
		hash_idx = working ? hash + ((i%3) * PIECE_SIZE) : PART_HASH_MAP_SIZE;
		old_value = s[hash_idx];			// read old value
		__syncthreads();
		s[hash_idx] = value;				// write new value
		__syncthreads();
		if(value.x == s[hash_idx].x)		// check for success
		{
			if(value.y != s[hash_idx].y)
				s[hash_idx] = old_value;
			else if(old_value.x == EMPTY_BUCKET_KEY)
				working = false;
			else {
				value = old_value;
				old_value = int2{EMPTY_BUCKET_KEY, EMPTY_BUCKET_KEY};
			}
		}
		__syncthreads();
	}

	if(working) // try to add to stash
	{
		hash_idx = atomicAdd(stashCnt, 1);
		if(hash_idx < DEFAULT_STASH_SIZE)
			stash[hash_idx] = value;
		else
			atomicAdd(failures, 1);
	}

	// COPY SHARED MEMORY TO HASH MAP
	__syncthreads();
	if(idx2 < PART_HASH_MAP_SIZE)
		hashMap_part[idx2] = s[idx2];
	hashMap_part[idx] = s[idx];
}

bool fast_cuckooHash(
		const int2* values,
		const int in_size,
		int2* hashMap,
		const int bucket_cnt,
		Constants<2> bucket_constants,
		Constants<3> constants,
		int max_iters)
{
	const int block_size = FAST_CUCKOO_HASH_BLOCK_SIZE;
	unsigned int* starts;
	unsigned int* counts;
	int2* buckets;
	int* d_failure, * d_cnt;
	int h_failure, h_cnt;
	const int steam_no = bucket_cnt < MAX_STREAM_NO ? bucket_cnt : MAX_STREAM_NO;

	// CREATE STREAMS
	hipStream_t* streams = new hipStream_t[steam_no];
	for(int i=0; i<steam_no; i++)
		CUDA_CALL( hipStreamCreate(&streams[i]) );

	// ALLOCATE MEMORY
	CUDA_CALL( hipMalloc((void**)&starts, bucket_cnt*sizeof(unsigned int)) );
	CUDA_CALL( hipMemset(starts, 0, bucket_cnt*sizeof(unsigned int)) );

	CUDA_CALL( hipMalloc((void**)&counts, bucket_cnt*sizeof(unsigned int)) );
	CUDA_CALL( hipMemset(counts, 0, bucket_cnt*sizeof(unsigned int)) );

	CUDA_CALL( hipMalloc((void**)&buckets, in_size*sizeof(int2)) );
	CUDA_CALL( hipMemset(buckets, 0xff, in_size*sizeof(int2)) );

	CUDA_CALL( hipMalloc((void**)&d_failure, sizeof(int)) );
	CUDA_CALL( hipMemset(d_failure, 0, sizeof(int)) );

	CUDA_CALL( hipMalloc((void**)&d_cnt, sizeof(int)) );
	CUDA_CALL( hipMemset(d_cnt, 0, sizeof(int)) );

	bool splitResult = splitToBuckets(
			values, in_size, bucket_constants, bucket_cnt,
			block_size, starts, counts, buckets);

//	printData(buckets, in_size, "Buckets: ");

	int2* stash = hashMap + (bucket_cnt * PART_HASH_MAP_SIZE);

	if(splitResult)
	{
		for(int i=0; i<bucket_cnt; i++)
		{
			insertKernel<<<1, block_size, 0, 0>>>(//streams[i%steam_no]>>>(
					buckets, starts, counts, i, hashMap, stash, d_cnt, constants, d_failure);
		}
		hipDeviceSynchronize();
		CUDA_CALL( hipMemcpy(&h_failure, d_failure, sizeof(int), hipMemcpyDeviceToHost) );
		CUDA_CALL( hipMemcpy(&h_cnt, d_cnt, sizeof(int), hipMemcpyDeviceToHost) );
	} else return true;

	// FREE MEMORY
	CUDA_CALL( hipFree(starts) );
	CUDA_CALL( hipFree(counts) );
	CUDA_CALL( hipFree(buckets) );
	CUDA_CALL( hipFree(d_failure) );
	CUDA_CALL( hipFree(d_cnt) );
	for(int i=0; i<steam_no; i++)
		CUDA_CALL( hipStreamDestroy(streams[i]) );
	delete streams;

	printf("FAILURES NO: %d\n", h_failure);
	printf("STASH CNT: %d\n", h_cnt);
	printf("BUCKET CNT: %d\n", bucket_cnt);

	return h_failure;
}

__global__ void toInt2Kernel(const int* keys, const int size, int2* out)
{
	unsigned idx = threadIdx.x + blockIdx.x * blockDim.x +
				blockIdx.y * blockDim.x * gridDim.x;

	if(idx >= size) return;
	out[idx].x = keys[idx];
	out[idx].y = EMPTY_BUCKET_KEY;
}

__global__ void retrieveKernel(
		int2* values,
		int2* hashMap,
		int2* stash,
		int size,
		int bucket_cnt,
		Constants<3> constants,
		Constants<2> bucket_constants)
{
	unsigned idx = threadIdx.x + blockIdx.x * blockDim.x +
					blockIdx.y * blockDim.x * gridDim.x;
	if(idx >= size) return;

	int key = values[idx].x;

    unsigned hash = bucketHashFunction(
			bucket_constants.values[0], bucket_constants.values[1], key, bucket_cnt);
    unsigned bucket_start = hash * PART_HASH_MAP_SIZE;
    unsigned hash_idx;
    int2 entry;

	for(int i = 0; i < 3; i++)
	{
		hash = hashFunction(constants.values[i%3], key, PIECE_SIZE);
		hash_idx = hash + ((i%3) * PIECE_SIZE) + bucket_start;
		entry = hashMap[hash_idx];
		if(entry.x == key) break;
	}

	if(entry.x != key) // check stash
	{
		for(int i=0; i < DEFAULT_STASH_SIZE; i++)
			if(stash[i].x == key)
			{
				entry = stash[i];
				hash_idx = -i;
				break;
			}
	}

	values[idx] = entry;
}

int2* fast_cuckooRetrieve(
		const int* keys,
		const int size,
		int2* hashMap,
		const int bucket_cnt,
		const Constants<2> bucket_constants,
		const Constants<3> constants)
{
	auto grid = CuckooHash<2>::GetGrid(size);
	int blockSize = CuckooHash<2>::DEFAULT_BLOCK_SIZE;

//	printHashMap(hashMap, bucket_cnt*PART_HASH_MAP_SIZE, "Hash Map:");
//	printHashMap(hashMap + bucket_cnt*PART_HASH_MAP_SIZE, DEFAULT_STASH_SIZE, "Stash:");

	// ALLOCATE MEMORY
	int2 *result;
	int2 *stash = hashMap + (bucket_cnt * PART_HASH_MAP_SIZE);
	CUDA_CALL( hipMalloc((void**)&result, size*sizeof(int2)) );
	CUDA_CALL( hipMemset(result, 0xff, size*sizeof(int2)) );

	// SPLIT TO BUCKETS
	toInt2Kernel<<<grid, blockSize>>>(keys, size, result);
	hipDeviceSynchronize();

	retrieveKernel<<<grid, blockSize>>>(
			result, hashMap, stash, size, bucket_cnt, constants, bucket_constants);
	hipDeviceSynchronize();

	CudaCheckError();
	return result;
}
